
#include <hip/hip_runtime.h>
#include <iostream>
#include <utility>
#include <algorithm>
#include <sys/time.h>

using namespace std;

const int INF = 1e9 + 7;

bool cmp(pair<int, int> a, pair<int, int> b) {
    if(a.first == b.first) {
        return a.second < b.second;
    } else {
        return a.first < b.first;
    }
}

__global__ 
void reduceMin(int n ,pair<int, int> *min_edge, bool *visited, int* minval, int* idxmin)
{
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int localmin = INF;
    int localidxmin = -1;
    for(int j = index; j < n; j += stride) {
        if(visited[j] == 0 && (localidxmin == -1 || min_edge[j].first < min_edge[localidxmin].first)) {
            localidxmin = j;
            localmin = min_edge[j].first;
        }
    }
 
    atomicMin(minval, localmin);
  
    __syncthreads();
    
    if(*minval == localmin) {
        *idxmin = localidxmin;
    }
}

int main(){
    
    int n;
    
    cin >> n;

    int *adj, *idxmin, *minval;
    pair<int, int> *min_edge, *result;
    bool *visited;

    hipMallocManaged(&adj, n * n * sizeof(int));
    hipMallocManaged(&idxmin, sizeof(int));
    hipMallocManaged(&minval, sizeof(int));
    hipMallocManaged(&min_edge, n * sizeof(pair<int, int>));
    hipMallocManaged(&result, n * sizeof(pair<int, int>));
    hipMallocManaged(&visited, n * sizeof(bool));


    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cin >> adj[i * n + j];
            // akses tidak bisa [][]. harus [], maka diflatten
            if(adj[i * n + j] == -1) adj[i * n + j] = INF;
        }

        visited[i] = 0;
        // first: weight, second: terhubung sama apa
        min_edge[i].first = INF;
        min_edge[i].second = -1;
    }

    int total_weight = 0;
    min_edge[0].first = 0;

    int cur = 0;

    struct timeval stop, start;
    gettimeofday(&start, NULL);
    for(int i = 0; i < n; i++) {
        int blockSize = 256;
        
        int numBlocks = (n + blockSize - 1) / blockSize;

        *idxmin = -1;
        *minval = INF;
        reduceMin<<<numBlocks, blockSize>>>(n, min_edge, visited, minval, idxmin);

        hipDeviceSynchronize();

        int t = *idxmin;
        visited[t] = 1;
        total_weight += min_edge[t].first;
        if(min_edge[t].second != -1) {
            result[cur].first = min(t, min_edge[t].second);
            result[cur].second = max(t, min_edge[t].second);
            cur++;
        }
        //cout << *idxmin << " this is " << *minval << '\n';
        for(int to = 0; to < n; to++) {
            if(adj[t * n + to] < min_edge[to].first) {
                min_edge[to].first = adj[t * n + to];
                min_edge[to].second = t;
            }
            //cout << min_edge[to].first << " - " << min_edge[to].second << '\n';

        }
    }
    gettimeofday(&stop, NULL);

    sort(result, result + cur, cmp);

    cout << total_weight << '\n';

    for(int i = 0; i < cur; i++) {
        cout << result[i].first << '-' << result[i].second << '\n';
    }

    cout << "Waktu Eksekusi: " << (stop.tv_sec - start.tv_sec) * 1000 + (stop.tv_usec - start.tv_usec) / 1000 << " ms\n";
    hipFree(adj);
    hipFree(idxmin);
    hipFree(minval);
    hipFree(min_edge);
    hipFree(result);
    hipFree(visited);
    
    return 0;
}